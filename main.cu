#include "Operators.cuh"
#include "Reduce.cuh"
#include "Scan.cuh"

#include <ctime>
#include <cstdlib>
#include <iostream>
using namespace std;

#define TYPE int

int main() {
	srand(time(0));
	size_t n = 1 << 24;
	TYPE* arr = new TYPE[n];
	for (size_t i = 0; i < n; ++i) {
		arr[i] = static_cast<TYPE>(20.0 * rand() / RAND_MAX - 10.0);
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// hipEventRecord(start);
	// hipEventSynchronize(start);

	// TYPE reduce_cpu = ReduceCPU::reduce(arr, n, Operators::add<TYPE>());
	
	// hipEventRecord(stop);
	// hipEventSynchronize(stop);
	// float cpu_time = 0.0f;
	// hipEventElapsedTime(&cpu_time, start, stop);

	// cout << "Reduce CPU: " << reduce_cpu << endl;
	// cout << "Execution time: " << cpu_time << " ms" << endl;

	// hipEventRecord(start);
	// hipEventSynchronize(start);

	// TYPE reduce_gpu = ReduceGPU::reduce(arr, n, Operators::add<TYPE>());

	// hipEventRecord(stop);
	// hipEventSynchronize(stop);
	// float gpu_time = 0.0f;
	// hipEventElapsedTime(&gpu_time, start, stop);

	// cout << "Reduce GPU: " << reduce_gpu << endl;
	// cout << "Execution time: " << gpu_time << " ms" << endl;

	hipEventRecord(start);
	hipEventSynchronize(start);

	TYPE* exc_scan_cpu = ScanCPU::exclusive_scan(arr, n, Operators::add<TYPE>());

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float cpu_time = 0.0f;
	hipEventElapsedTime(&cpu_time, start, stop);

	cout << "Exclusive Scan CPU: ";
	for (size_t i = 0; i < n; i += n >> 3)
		cout << exc_scan_cpu[i] << " ";
	cout << endl;
	cout << "Execution time: " << cpu_time << " ms" << endl;

	hipEventRecord(start);
	hipEventSynchronize(start);

	TYPE* exc_scan_gpu = ScanGPUEfficient::exclusive_scan(arr, n, Operators::add<TYPE>());

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float gpu_time = 0.0f;
	hipEventElapsedTime(&gpu_time, start, stop);

	cout << "Exclusive Scan GPU: ";
	for (size_t i = 0; i < n; i += n >> 3)
		cout << exc_scan_gpu[i] << " ";
	cout << endl;
	cout << "Execution time: " << gpu_time << " ms" << endl;

	delete [] exc_scan_cpu;
	delete [] exc_scan_gpu;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	delete [] arr;
}