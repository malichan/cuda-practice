#include "Operators.cuh"
#include "Reduce.cuh"
#include "Scan.cuh"

#include <ctime>
#include <cstdlib>
#include <iostream>
using namespace std;

#define TYPE float

int main() {
	srand(time(0));
	size_t n = 100000000;
	TYPE* arr = new TYPE[n];
	for (size_t i = 0; i < n; ++i) {
		arr[i] = static_cast<TYPE>(20.0 * rand() / RAND_MAX - 10.0);
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	hipEventSynchronize(start);

	TYPE reduce_cpu = ReduceCPU::reduce(arr, n, Operators::add<TYPE>());
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float cpu_time = 0.0f;
	hipEventElapsedTime(&cpu_time, start, stop);

	cout << "Reduce CPU: " << reduce_cpu << endl;
	cout << "Execution time: " << cpu_time << " ms" << endl;

	hipEventRecord(start);
	hipEventSynchronize(start);

	TYPE reduce_gpu = ReduceGPU::reduce(arr, n, Operators::add<TYPE>());

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float gpu_time = 0.0f;
	hipEventElapsedTime(&gpu_time, start, stop);

	cout << "Reduce GPU: " << reduce_gpu << endl;
	cout << "Execution time: " << gpu_time << " ms" << endl;

	// TYPE* inc_scan_cpu = ScanCPU::inclusive_scan(arr, n, Operators::add<TYPE>());
	// cout << "Inclusive Scan CPU: ";
	// for (size_t i = 0; i < n; i += n >> 3)
	// 	cout << inc_scan_cpu[i] << " ";
	// cout << endl;
	// TYPE* inc_scan_gpu = ScanGPUBlelloch::inclusive_scan(arr, n, Operators::add<TYPE>());
	// cout << "Inclusive Scan GPU: ";
	// for (size_t i = 0; i < n; i += n >> 3)
	// 	cout << inc_scan_gpu[i] << " ";
	// cout << endl;

	// delete [] inc_scan_cpu;
	// delete [] inc_scan_gpu;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	delete [] arr;
}